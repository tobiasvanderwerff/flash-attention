#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <c10/cuda/CUDAException.h>

__device__ inline unsigned int cdiv(unsigned int a, unsigned int b) { return (a + b - 1) / b; }


template <int BLOCK_SIZE>
__global__ void softmax_kernel_2(float* out, const float* inp, int h, int w) {
    /* Softmax applied row-wise. 

    Replaces shared memory with warp-level shuffles.
    It also uses packed data structures.

    I.e. instead of thread coarsening by iterating over blocks:
    - reduce 2x per block using 32 sized warps (32*32=1024, which is assumed to be max block size)
        - or, when k is small, iterate over warps
    - use a packed data structure (float4?)

    Inspiration:
    - https://github.com/facebookincubator/AITemplate/wiki/How-to-write-a-fast-Softmax-CUDA-kernel%3F
    - https://developer.nvidia.com/blog/register-cache-warp-cuda/

    NB: be careful with register spilling (e.g. 25 int registers for a single thread is pushing it). Quote from second link:
    > the efficiency of the register cache is predicated on the availability of
    > spare registers. Otherwise, registers start spilling to global memory,
    > leading to a dramatic performance drop, as is the case for k=25 in Figure 6
    */

    // TODO
}

template <typename T>
__inline__ __device__ void warp_reduce_sum(T* val) {
    #pragma unroll
    for (int stride = 1; stride < warpSize; stride *= 2)
        val[0] += __shfl_xor_sync(0xffffffff, val[0], stride);
}

template <int BLOCK_SIZE>
__global__ void softmax_kernel(float* out, const float* inp, int h, int w) {
    /* Softmax applied row-wise. 

    softmax(x) = exp(x) / sum(exp(x))
    
    In this kernel, each block handles a single row.
    */

    __shared__ float shm[BLOCK_SIZE];

    const int tx = threadIdx.x;
    const int bx = blockIdx.x;

    // Calculate max value of the row
    float max_val = -INFINITY;
    for (int bi = 0; bi < cdiv(w, BLOCK_SIZE); ++bi) { // Thread coarsening
        int col = bi*BLOCK_SIZE + tx;
        shm[tx] = (col < w) ? inp[bx*w + col] : -INFINITY;

        __syncthreads();
        for (int stride = BLOCK_SIZE >> 1; stride >= 1; stride >>= 1) {
            if (tx < stride)
                shm[tx] = fmaxf(shm[tx], shm[tx + stride]);
            __syncthreads();
        }

        max_val = fmaxf(max_val, shm[0]);
    }

    float sum = 0.0f;
    for (int bi = 0; bi < cdiv(w, BLOCK_SIZE); ++bi) { // Thread coarsening
        // Calculate exponent element-wise
        int idx = bx*w + bi*BLOCK_SIZE + tx;
        if (bi*BLOCK_SIZE + tx < w) {
            float e = expf(inp[idx] - max_val);  // TODO: use __expf?
            out[idx] = e;
            shm[tx] = e;
        } else {
            shm[tx] = 0.0f;
        }

        // Calculate sum of exponents
        // Note: block size is assumed to be power of 2
        __syncthreads();
        for (int stride = BLOCK_SIZE >> 1; stride >= 1; stride >>= 1) {
            if (tx < stride)
                shm[tx] += shm[tx + stride];
            __syncthreads();
        }

        // Let all threads save the intermediate sum
        sum += shm[0];
        __syncthreads();
    }

    // Divide by exponent sum
    for (int bi = 0; bi < cdiv(w, BLOCK_SIZE); ++bi) { // Thread coarsening
        if (bi*BLOCK_SIZE + tx < w) 
            out[bx*w + bi*BLOCK_SIZE + tx] /= sum;
    }
}

template <int BLOCK_SIZE>
void launch_softmax_kernel(int gdim, int bdim, float* out, const float* inp, int h, int w) { 
    softmax_kernel<BLOCK_SIZE><<<gdim, bdim>>>(out, inp, h, w);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
}

template void launch_softmax_kernel<64>(int gdim, int bdim, float* out, const float* inp, int h, int w);
template void launch_softmax_kernel<128>(int gdim, int bdim, float* out, const float* inp, int h, int w);
template void launch_softmax_kernel<256>(int gdim, int bdim, float* out, const float* inp, int h, int w);
template void launch_softmax_kernel<512>(int gdim, int bdim, float* out, const float* inp, int h, int w);
template void launch_softmax_kernel<1024>(int gdim, int bdim, float* out, const float* inp, int h, int w);

