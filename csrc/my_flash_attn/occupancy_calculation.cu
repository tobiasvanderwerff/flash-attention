// A simple example to demonstrate the CUDA API for calculating launch
// parameters according to what leads to the highest occupancy.
// Adapted from https://developer.nvidia.com/blog/cuda-pro-tip-occupancy-api-simplifies-launch-configuration/
//
// To compile and run this code, run:
//    nvcc -o occupancy_calculation occupancy_calculation.cu
//    ./occupancy_calculation


#include <hip/hip_runtime.h>
#include "stdio.h"

__global__ void MyKernel(int *array, int arrayCount) 
{ 
  int idx = threadIdx.x + blockIdx.x * blockDim.x; 
  if (idx < arrayCount) 
  { 
    array[idx] *= array[idx]; 
  } 
} 

void launchMyKernel(int *array, int arrayCount) 
{ 
  int blockSize;   // The launch configurator returned block size 
  int minGridSize; // The minimum grid size needed to achieve the 
                   // maximum occupancy for a full device launch 
  int gridSize;    // The actual grid size needed, based on input size 

  hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize, 
                                      MyKernel, 0, 0); 
  // Round up according to array size 
  gridSize = (arrayCount + blockSize - 1) / blockSize; 

  MyKernel<<< gridSize, blockSize >>>(array, arrayCount); 

  hipDeviceSynchronize(); 

  // calculate theoretical occupancy
  int maxActiveBlocks;
  hipOccupancyMaxActiveBlocksPerMultiprocessor( &maxActiveBlocks, 
                                                 MyKernel, blockSize, 
                                                 0);

  int device;
  hipDeviceProp_t props;
  hipGetDevice(&device);
  hipGetDeviceProperties(&props, device);

  float occupancy = (maxActiveBlocks * blockSize / props.warpSize) / 
                    (float)(props.maxThreadsPerMultiProcessor / 
                            props.warpSize);

  printf("Launched blocks of size %d. Theoretical occupancy: %f\n", 
         blockSize, occupancy);
}


int main() {
    const int n = 1 << 24; // Adjust the data size for workload
    int *in;

    hipMallocManaged(&in, n * sizeof(int));

    for(int i = 0; i < n; ++i) {
        in[i] = i;
    }

    launchMyKernel(in, n);

    hipDeviceSynchronize();

    hipFree(in);

    return 0;
}
